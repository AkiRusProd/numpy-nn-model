
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <cstdlib>
#include <iostream>
#define DLLEXPORT extern "C" __declspec(dllexport)
using namespace std;

// CUDA kernel for adding bias to each column in the output matrix C
__global__ void addBiasKernel(float *C, const float *bias, int rowsNum, int colsNum) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < colsNum) {
        for (int i = 0; i < rowsNum; i++) {
            C[i * colsNum + idx] += bias[idx];
        }
    }
}

// CUDA kernel для вычисления суммы градиентов по bias
__global__ void sumBiasKernel(float *d_bias, const float *d_output, int rowsNum, int colsNum) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < colsNum) {
        float sum = 0.0f;
        for (int i = 0; i < rowsNum; i++) {
            sum += d_output[i * colsNum + idx]; // Суммируем значения по строкам
        }
        d_bias[idx] = sum; // Результат сохраняем в соответствующем bias
    }
}

// C(rowsNum, colsNum) = A(rowsNum, width) * B(width, colsNum) + bias
void blasMatMulWithBias(const float *A, const float *B, const float *bias, float *C, const int rowsNum, const int width, const int colsNum) {
    const float alf = 1.0f;
    const float bet = 0.0f;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Matrix multiplication: C = A * B
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, colsNum, rowsNum, width, &alf, B, colsNum, A, width, &bet, C, colsNum);

    // Add bias to each column of C
    // C(i, j) += bias(j)
    int threadsPerBlock = 256;  // You can experiment with this value
    int blocksPerGrid = (rowsNum + threadsPerBlock - 1) / threadsPerBlock;

    // Add bias to each row for each column (without race conditions)
    addBiasKernel<<<blocksPerGrid, threadsPerBlock>>>(C, bias, rowsNum, colsNum);

    hipDeviceSynchronize();  // Make sure the kernel execution finishes

    hipblasDestroy(handle);
}



DLLEXPORT void cudaLinearModuleForward(float *input, float *weights, float *bias, float *output, int inputRowsNum, int inputColsNum, int outputColsNum) {
    float *p_input, *p_weights, *p_bias, *p_output;

    // Device memory allocation
    hipMalloc((void**)&p_input, inputRowsNum * inputColsNum * sizeof(float));
    hipMalloc((void**)&p_weights, inputColsNum * outputColsNum * sizeof(float));
    hipMalloc((void**)&p_bias, outputColsNum * sizeof(float));
    hipMalloc((void**)&p_output, inputRowsNum * outputColsNum * sizeof(float));

    // Copy data to device
    hipMemcpy(p_input, input, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(p_weights, weights, inputColsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(p_bias, bias, outputColsNum * sizeof(float), hipMemcpyHostToDevice);

    // Perform linear layer operation: output = input * weights + bias
    blasMatMulWithBias(p_input, p_weights, p_bias, p_output, inputRowsNum, inputColsNum, outputColsNum);

    // Copy result back to host
    hipMemcpy(output, p_output, inputRowsNum * outputColsNum * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(p_input);
    hipFree(p_weights);
    hipFree(p_bias);
    hipFree(p_output);
}



DLLEXPORT void cudaLinearModuleBackward(
    float *input, float *weights, float *d_output, 
    float *d_input, float *d_weights, float *d_bias, 
    int inputRowsNum, int inputColsNum, int outputColsNum) {

    float *p_input, *p_weights, *p_d_output, *p_d_input, *p_d_weights, *p_d_bias;

    // Device memory allocation
    hipMalloc((void**)&p_input, inputRowsNum * inputColsNum * sizeof(float));
    hipMalloc((void**)&p_weights, inputColsNum * outputColsNum * sizeof(float));
    hipMalloc((void**)&p_d_output, inputRowsNum * outputColsNum * sizeof(float));
    hipMalloc((void**)&p_d_input, inputRowsNum * inputColsNum * sizeof(float));
    hipMalloc((void**)&p_d_weights, inputColsNum * outputColsNum * sizeof(float));
    hipMalloc((void**)&p_d_bias, outputColsNum * sizeof(float));

    // Copy data to device
    hipMemcpy(p_input, input, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(p_weights, weights, inputColsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(p_d_output, d_output, inputRowsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alf = 1.0f;
    const float bet = 0.0f;

    // Compute d_input = d_output * W^T
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, inputColsNum, inputRowsNum, outputColsNum,
                &alf, p_weights, outputColsNum, p_d_output, outputColsNum, &bet, p_d_input, inputColsNum);

    // Compute d_weights = A^T * d_output
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, outputColsNum, inputColsNum, inputRowsNum,
                &alf, p_d_output, outputColsNum, p_input, inputColsNum, &bet, p_d_weights, outputColsNum);

    // Compute d_bias = sum(d_output, axis=0)
    // Use custom kernel to sum along rows
    int threadsPerBlock = 256;
    int blocksPerGrid = (outputColsNum + threadsPerBlock - 1) / threadsPerBlock;
    sumBiasKernel<<<blocksPerGrid, threadsPerBlock>>>(p_d_bias, p_d_output, inputRowsNum, outputColsNum);

    hipDeviceSynchronize();

    // Copy results back to host
    hipMemcpy(d_input, p_d_input, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(d_weights, p_d_weights, inputColsNum * outputColsNum * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(d_bias, p_d_bias, outputColsNum * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(p_input);
    hipFree(p_weights);
    hipFree(p_d_output);
    hipFree(p_d_input);
    hipFree(p_d_weights);
    hipFree(p_d_bias);

    hipblasDestroy(handle);
}
