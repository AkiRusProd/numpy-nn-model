
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <hipblaslt.h>
#include <cstdlib>
#include <iostream>

#ifdef _WIN32
#define DLLEXPORT extern "C" __declspec(dllexport)
#else
#define DLLEXPORT extern "C"
#endif

using namespace std;

// CUDA kernel for summing gradients for bias
__global__ void sumBiasKernel(float *d_bias, const float *d_output, int rowsNum, int colsNum) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < colsNum) {
        float sum = 0.0f;
        for (int i = 0; i < rowsNum; i++) {
            sum += d_output[i * colsNum + idx];
        }
        d_bias[idx] = sum;
    }
}

// Global cacheable forward pointers to GPU memory
static float *p_input_fw = nullptr;
static float *p_weights_fw = nullptr;
static float *p_output_fw = nullptr;
static float *p_bias_fw = nullptr;

// Current sizes of cached tensors in forward
static int current_input_fw_rows = 0;
static int current_input_fw_cols = 0;
static int current_weights_fw_rows = 0;
static int current_weights_fw_cols = 0;
static int current_output_fw_rows = 0;
static int current_output_fw_cols = 0;
static int current_bias_fw_size = 0;


// Global cacheable backward pointers to GPU memory
static float *p_input_bw = nullptr;       
static float *p_weights_bw = nullptr;   
static float *p_d_output_bw = nullptr;
static float *p_d_input_bw = nullptr;
static float *p_d_weights_bw = nullptr;
static float *p_d_bias_bw = nullptr;

// Current sizes of cached tensors in backward
static int current_input_bw_rows = 0;
static int current_input_bw_cols = 0;
static int current_weights_bw_rows = 0;
static int current_weights_bw_cols = 0;
static int current_d_output_bw_rows = 0;
static int current_d_output_bw_cols = 0;
static int current_d_input_bw_rows = 0;
static int current_d_input_bw_cols = 0;
static int current_d_weights_bw_rows = 0;
static int current_d_weights_bw_cols = 0;
static int current_d_bias_bw_size = 0;

// Cached handle for cuBLAS
static hipblasLtHandle_t cublaslt_handle = nullptr;

static const size_t cublaslt_workspace_size = 4 * 1024 * 1024; // 4MB
static void *cublaslt_workspace = nullptr;




// Helper function for matrix multiplication with cuBLASLt
void matmul_cublaslt(
    float* C, const float* A, const float* B,
    int m, int n, int k,
    bool transA, bool transB,
    float alpha = 1.0f, float beta = 0.0f,
    const float* bias = nullptr
) {
    hipblasLtMatmulDesc_t operationDesc;
    hipblasLtMatrixLayout_t Adesc, Bdesc, Cdesc;
    hipblasLtMatmulPreference_t preference;

    // Create descriptors

    hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, 
                                  &opA, sizeof(opA));
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, 
                                  &opB, sizeof(opB));

    // Configure epilogue for bias in forward mode
    if (bias != nullptr) {
        hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_BIAS;
        hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
        hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    }

    // Create matrix layouts
    hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, 
                              transA ? k : m, transA ? m : k, 
                              transA ? k : m);
    hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, 
                              transB ? n : k, transB ? k : n, 
                              transB ? n : k);
    hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, m, n, m);

    // Set preference
    hipblasLtMatmulPreferenceCreate(&preference);
    hipblasLtMatmulPreferenceSetAttribute(preference, 
        HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, 
        &cublaslt_workspace_size, sizeof(cublaslt_workspace_size));

    // Find heuristic
    hipblasLtMatmulHeuristicResult_t heuristic;
    int returnedResults = 0;
    hipblasLtMatmulAlgoGetHeuristic(
        cublaslt_handle, operationDesc, Adesc, Bdesc, Cdesc, Cdesc,
        preference, 1, &heuristic, &returnedResults
    );

    if (returnedResults == 0) {
        printf("cuBLASLt heuristic not found!\n");
        exit(EXIT_FAILURE);
    }

    // Run matmul
    hipblasLtMatmul(
        cublaslt_handle, operationDesc,
        &alpha, A, Adesc, B, Bdesc,
        &beta, C, Cdesc, C, Cdesc,
        &heuristic.algo, cublaslt_workspace, cublaslt_workspace_size, 0
    );

    // Cleanup
    hipblasLtMatmulPreferenceDestroy(preference);
    hipblasLtMatmulDescDestroy(operationDesc);
    hipblasLtMatrixLayoutDestroy(Adesc);
    hipblasLtMatrixLayoutDestroy(Bdesc);
    hipblasLtMatrixLayoutDestroy(Cdesc);
}


extern "C" {
    DLLEXPORT void cudaLinearModuleForward(float *input, float *weights, float *bias, float *output, int inputRowsNum, int inputColsNum, int outputColsNum) {

        // check alignment (some modes work unaligned but it always best to be aligned for performance)
        if(((uintptr_t)input % 16) != 0 || ((uintptr_t)weights % 16) != 0 || ((uintptr_t)bias % 16) != 0 || ((uintptr_t)output % 16) != 0) {
            printf("All cuBLASLt pointers must be aligned!\n");
            exit(EXIT_FAILURE);
        }

        // Check and allocate memory for p_input
        if (p_input_fw == nullptr || current_input_fw_rows != inputRowsNum || current_input_fw_cols != inputColsNum) {
            hipFree(p_input_fw);
            hipMalloc(&p_input_fw, inputRowsNum * inputColsNum * sizeof(float));
            current_input_fw_rows = inputRowsNum;
            current_input_fw_cols = inputColsNum;
        }
    
        // Check and allocate memory for p_weights
        if (p_weights_fw == nullptr || current_weights_fw_rows != inputColsNum || current_weights_fw_cols != outputColsNum) {
            hipFree(p_weights_fw);
            hipMalloc(&p_weights_fw, inputColsNum * outputColsNum * sizeof(float));
            current_weights_fw_rows = inputColsNum;
            current_weights_fw_cols = outputColsNum;
        }
    
        // Check and allocate memory for p_output
        if (p_output_fw == nullptr || current_output_fw_rows != inputRowsNum || current_output_fw_cols != outputColsNum) {
            hipFree(p_output_fw);
            hipMalloc(&p_output_fw, inputRowsNum * outputColsNum * sizeof(float));
            current_output_fw_rows = inputRowsNum;
            current_output_fw_cols = outputColsNum;
        }
    
        // Handle bias
        if (bias != nullptr) {
            if (p_bias_fw == nullptr || current_bias_fw_size != outputColsNum) {
                hipFree(p_bias_fw);
                hipMalloc(&p_bias_fw, outputColsNum * sizeof(float));
                current_bias_fw_size = outputColsNum;
            }
            hipMemcpy(p_bias_fw, bias, outputColsNum * sizeof(float), hipMemcpyHostToDevice);
        } else {
            if (p_bias_fw != nullptr) {
                hipFree(p_bias_fw);
                p_bias_fw = nullptr; // Explicitly set to nullptr
                current_bias_fw_size = 0;
            }
        }
    
        // Copy data to device
        hipMemcpy(p_input_fw, input, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(p_weights_fw, weights, inputColsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);

        if (cublaslt_handle == nullptr) {
            hipblasLtCreate(&cublaslt_handle);
            // Allocate workspace once
            if (cublaslt_workspace == nullptr) {
                hipMalloc(&cublaslt_workspace, cublaslt_workspace_size);
            }
        }

        matmul_cublaslt(
            p_output_fw, p_weights_fw, p_input_fw,     // Result, Weights, Input
            outputColsNum, inputRowsNum, inputColsNum, // m, n, k
            true,           // Transpose Weights (transA)
            false,          // Do not transpose Input (transB)
            1.0f,           // alpha
            0.0f,           // beta
            p_bias_fw       // bias (if not null)
        );

        hipMemcpy(output, p_output_fw, inputRowsNum * outputColsNum * sizeof(float), hipMemcpyDeviceToHost);

        
    }
    
    DLLEXPORT void cudaLinearModuleBackward(
        float *input, float *weights, float *d_output, 
        float *d_input, float *d_weights, float *d_bias, 
        int inputRowsNum, int inputColsNum, int outputColsNum) {

        if(((uintptr_t)input % 16) != 0 || ((uintptr_t)weights % 16) != 0 || ((uintptr_t)d_output % 16) != 0 || 
            ((uintptr_t)d_input % 16) != 0 || ((uintptr_t)d_weights % 16) != 0 || ((uintptr_t)d_bias % 16) != 0) 
         {
             printf("All cuBLASLt pointers must be aligned!\n");
             exit(EXIT_FAILURE);
         }
    
    
        // Check and allocate memory for p_input
        if (p_input_bw == nullptr || current_input_bw_rows != inputRowsNum || current_input_bw_cols != inputColsNum) {
            hipFree(p_input_bw);
            hipMalloc(&p_input_bw, inputRowsNum * inputColsNum * sizeof(float));
            current_input_bw_rows = inputRowsNum;
            current_input_bw_cols = inputColsNum;
        }
    
        // Check and allocate memory for p_weights
        if (p_weights_bw == nullptr || current_weights_bw_rows != inputColsNum || current_weights_bw_cols != outputColsNum) {
            hipFree(p_weights_bw);
            hipMalloc(&p_weights_bw, inputColsNum * outputColsNum * sizeof(float));
            current_weights_bw_rows = inputColsNum;
            current_weights_bw_cols = outputColsNum;
        }
    
        // Check and allocate memory for p_d_output
        if (p_d_output_bw == nullptr || current_d_output_bw_rows != inputRowsNum || current_d_output_bw_cols != outputColsNum) {
            hipFree(p_d_output_bw);
            hipMalloc(&p_d_output_bw, inputRowsNum * outputColsNum * sizeof(float));
            current_d_output_bw_rows = inputRowsNum;
            current_d_output_bw_cols = outputColsNum;
        }
    
        // Check and allocate memory for p_d_input
        if (p_d_input_bw == nullptr || current_d_input_bw_rows != inputRowsNum || current_d_input_bw_cols != inputColsNum) {
            hipFree(p_d_input_bw);
            hipMalloc(&p_d_input_bw, inputRowsNum * inputColsNum * sizeof(float));
            current_d_input_bw_rows = inputRowsNum;
            current_d_input_bw_cols = inputColsNum;
        }
    
        // Check and allocate memory for p_d_weights
        if (p_d_weights_bw == nullptr || current_d_weights_bw_rows != inputColsNum || current_d_weights_bw_cols != outputColsNum) {
            hipFree(p_d_weights_bw);
            hipMalloc(&p_d_weights_bw, inputColsNum * outputColsNum * sizeof(float));
            current_d_weights_bw_rows = inputColsNum;
            current_d_weights_bw_cols = outputColsNum;
        }
    
        // Check and allocate memory for p_d_bias
        if (d_bias != nullptr) {
            if (p_d_bias_bw == nullptr || current_d_bias_bw_size != outputColsNum) {
                hipFree(p_d_bias_bw);
                hipMalloc(&p_d_bias_bw, outputColsNum * sizeof(float));
                current_d_bias_bw_size = outputColsNum;
            }
        } else {
            if (p_d_bias_bw != nullptr) {
                hipFree(p_d_bias_bw);
                p_d_bias_bw = nullptr;
                current_d_bias_bw_size = 0;
            }
        }
    
        // Copy data to device
        hipMemcpy(p_input_bw, input, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(p_weights_bw, weights, inputColsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(p_d_output_bw, d_output, inputRowsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);
    
        // Initialize cuBLAS and cuBLASLt
        if (!cublaslt_handle) {
            hipblasLtCreate(&cublaslt_handle);
            hipMalloc(&cublaslt_workspace, cublaslt_workspace_size);
        }

        // Copy data to device (same as before)
        hipMemcpy(p_input_bw, input, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(p_weights_bw, weights, inputColsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(p_d_output_bw, d_output, inputRowsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);

        // Compute d_input = d_output * W^T
        matmul_cublaslt(
            p_d_input_bw, p_weights_bw, p_d_output_bw,  // Grad Input, Weights, Grad Output
            inputColsNum, inputRowsNum,  outputColsNum, // m, n, k
            false,
            false
        );

        // Compute d_weights = X^T * d_output
        matmul_cublaslt(
            p_d_weights_bw, p_input_bw, p_d_output_bw, // Grad Weights, Input, Grad Output
            inputColsNum, outputColsNum, inputRowsNum, // m, n, k
            false,
            true
        );

        // Compute d_bias = sum(d_output, axis=0)
        if (d_bias != nullptr && p_d_bias_bw != nullptr) {
            int threads = 256;
            int blocks = (outputColsNum + threads - 1) / threads;
            sumBiasKernel<<<blocks, threads>>>(
                p_d_bias_bw, p_d_output_bw, inputRowsNum, outputColsNum
            );
        }

        // Copy results back (same as before)
        hipMemcpy(d_input, p_d_input_bw, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(d_weights, p_d_weights_bw, inputColsNum * outputColsNum * sizeof(float), hipMemcpyDeviceToHost);
        if (d_bias != nullptr) {
            hipMemcpy(d_bias, p_d_bias_bw, outputColsNum * sizeof(float), hipMemcpyDeviceToHost);
        }
    }

    DLLEXPORT void cleanupCudaMemory() {
        // Freeing forward caches
        hipFree(p_input_fw); p_input_fw = nullptr;
        hipFree(p_weights_fw); p_weights_fw = nullptr;
        hipFree(p_output_fw); p_output_fw = nullptr;
        hipFree(p_bias_fw); p_bias_fw = nullptr;
        current_input_fw_rows = current_input_fw_cols = 0;
        current_weights_fw_rows = current_weights_fw_cols = 0;
        current_output_fw_rows = current_output_fw_cols = 0;
        current_bias_fw_size = 0;

        // Freeing backward caches
        hipFree(p_input_bw); p_input_bw = nullptr;
        hipFree(p_weights_bw); p_weights_bw = nullptr;
        hipFree(p_d_output_bw); p_d_output_bw = nullptr;
        hipFree(p_d_input_bw); p_d_input_bw = nullptr;
        hipFree(p_d_weights_bw); p_d_weights_bw = nullptr;
        hipFree(p_d_bias_bw); p_d_bias_bw = nullptr;
        current_input_bw_rows = current_input_bw_cols = 0;
        current_weights_bw_rows = current_weights_bw_cols = 0;
        current_d_output_bw_rows = current_d_output_bw_cols = 0;
        current_d_input_bw_rows = current_d_input_bw_cols = 0;
        current_d_weights_bw_rows = current_d_weights_bw_cols = 0;
        current_d_bias_bw_size = 0;

        // Destroying cuBLASlt handle
        if (cublaslt_handle != nullptr) {
            hipblasLtDestroy(cublaslt_handle);
            cublaslt_handle = nullptr;
        }
    }
}
