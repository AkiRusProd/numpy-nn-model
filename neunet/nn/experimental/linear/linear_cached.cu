
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <cstdlib>
#include <iostream>

#ifdef _WIN32
#define DLLEXPORT extern "C" __declspec(dllexport)
#else
#define DLLEXPORT extern "C"
#endif

using namespace std;

// CUDA kernel for adding bias to each column in the output matrix C
__global__ void addBiasKernel(float *C, const float *bias, int rowsNum, int colsNum) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < rowsNum && col < colsNum) {
         if (bias != nullptr) {
            C[row * colsNum + col] += bias[col];
        }
    }
}

// CUDA kernel for summing gradients for bias
__global__ void sumBiasKernel(float *d_bias, const float *d_output, int rowsNum, int colsNum) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < colsNum) {
        float sum = 0.0f;
        for (int i = 0; i < rowsNum; i++) {
            sum += d_output[i * colsNum + idx];
        }
        d_bias[idx] = sum;
    }
}

// Global cacheable forward pointers to GPU memory
static float *p_input_fw = nullptr;
static float *p_weights_fw = nullptr;
static float *p_output_fw = nullptr;
static float *p_bias_fw = nullptr;

// Current sizes of cached tensors in forward
static int current_input_fw_rows = 0;
static int current_input_fw_cols = 0;
static int current_weights_fw_rows = 0;
static int current_weights_fw_cols = 0;
static int current_output_fw_rows = 0;
static int current_output_fw_cols = 0;
static int current_bias_fw_size = 0;


// Global cacheable backward pointers to GPU memory
static float *p_input_bw = nullptr;       
static float *p_weights_bw = nullptr;   
static float *p_d_output_bw = nullptr;
static float *p_d_input_bw = nullptr;
static float *p_d_weights_bw = nullptr;
static float *p_d_bias_bw = nullptr;

// Current sizes of cached tensors in backward
static int current_input_bw_rows = 0;
static int current_input_bw_cols = 0;
static int current_weights_bw_rows = 0;
static int current_weights_bw_cols = 0;
static int current_d_output_bw_rows = 0;
static int current_d_output_bw_cols = 0;
static int current_d_input_bw_rows = 0;
static int current_d_input_bw_cols = 0;
static int current_d_weights_bw_rows = 0;
static int current_d_weights_bw_cols = 0;
static int current_d_bias_bw_size = 0;

// Cached handle for cuBLAS
static hipblasHandle_t cublas_handle = nullptr;


extern "C" {
    DLLEXPORT void cudaLinearModuleForward(float *input, float *weights, float *bias, float *output, int inputRowsNum, int inputColsNum, int outputColsNum) {

        // Check and allocate memory for p_input
        if (p_input_fw == nullptr || current_input_fw_rows != inputRowsNum || current_input_fw_cols != inputColsNum) {
            hipFree(p_input_fw);
            hipMalloc(&p_input_fw, inputRowsNum * inputColsNum * sizeof(float));
            current_input_fw_rows = inputRowsNum;
            current_input_fw_cols = inputColsNum;
        }
    
        // Check and allocate memory for p_weights
        if (p_weights_fw == nullptr || current_weights_fw_rows != inputColsNum || current_weights_fw_cols != outputColsNum) {
            hipFree(p_weights_fw);
            hipMalloc(&p_weights_fw, inputColsNum * outputColsNum * sizeof(float));
            current_weights_fw_rows = inputColsNum;
            current_weights_fw_cols = outputColsNum;
        }
    
        // Check and allocate memory for p_output
        if (p_output_fw == nullptr || current_output_fw_rows != inputRowsNum || current_output_fw_cols != outputColsNum) {
            hipFree(p_output_fw);
            hipMalloc(&p_output_fw, inputRowsNum * outputColsNum * sizeof(float));
            current_output_fw_rows = inputRowsNum;
            current_output_fw_cols = outputColsNum;
        }
    
        // Handle bias
        if (bias != nullptr) {
            if (p_bias_fw == nullptr || current_bias_fw_size != outputColsNum) {
                hipFree(p_bias_fw);
                hipMalloc(&p_bias_fw, outputColsNum * sizeof(float));
                current_bias_fw_size = outputColsNum;
            }
            hipMemcpy(p_bias_fw, bias, outputColsNum * sizeof(float), hipMemcpyHostToDevice);
        } else {
            if (p_bias_fw != nullptr) {
                hipFree(p_bias_fw);
                p_bias_fw = nullptr; // Explicitly set to nullptr
                current_bias_fw_size = 0;
            }
        }
    
        // Copy data to device
        hipMemcpy(p_input_fw, input, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(p_weights_fw, weights, inputColsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);
    
        // Perform linear layer operation: output = input * weights + bias
        // C(rowsNum, colsNum) = A(rowsNum, width) * B(width, colsNum) + bias
        if (cublas_handle == nullptr) {
            hipblasCreate(&cublas_handle);
        }

        const float alf = 1.0f;
        const float bet = 0.0f;
    
        // matrix multiplication: C = A * B^T
        hipblasSgemm(cublas_handle,
            HIPBLAS_OP_T,   // Transpose A (A is originally inputRowsNum x inputColsNum in row-major, becomes inputColsNum x inputRowsNum)
            HIPBLAS_OP_N,   // No transpose B (B is outputColsNum x inputColsNum in row-major)
            outputColsNum, // m: rows of op(A) and C (outputColsNum)
            inputRowsNum,  // n: columns of op(B) and C (inputRowsNum)
            inputColsNum,  // k: columns of op(A) and rows of op(B) (inputColsNum)
            &alf,
            p_weights_fw,     // A: originally B (outputColsNum x inputColsNum in row-major)
            inputColsNum,  // lda: leading dimension of A (inputColsNum, since row-major)
            p_input_fw,       // B: originally A (inputRowsNum x inputColsNum in row-major)
            inputColsNum,  // ldb: leading dimension of B (inputColsNum, since row-major)
            &bet,
            p_output_fw,      // C: result matrix (inputRowsNum x outputColsNum in row-major)
            outputColsNum);// ldc: leading dimension of C (outputColsNum, since row-major)

        // Add bias to each column of C
        if (bias != nullptr) {
            // C(i, j) += bias(j)
            dim3 threadsPerBlock(16, 16);
            dim3 blocksPerGrid((inputRowsNum + 15) / 16, (outputColsNum + 15) / 16);
            // Add bias to each row for each column (without race conditions)
            addBiasKernel<<<blocksPerGrid, threadsPerBlock>>>(p_output_fw, p_bias_fw, inputRowsNum, outputColsNum);
            hipDeviceSynchronize();
        }
    
        // Copy result back to host
        hipMemcpy(output, p_output_fw, inputRowsNum * outputColsNum * sizeof(float), hipMemcpyDeviceToHost);
    }
    
    DLLEXPORT void cudaLinearModuleBackward(
        float *input, float *weights, float *d_output, 
        float *d_input, float *d_weights, float *d_bias, 
        int inputRowsNum, int inputColsNum, int outputColsNum) {
    
        // Check and allocate memory for p_input
        if (p_input_bw == nullptr || current_input_bw_rows != inputRowsNum || current_input_bw_cols != inputColsNum) {
            hipFree(p_input_bw);
            hipMalloc(&p_input_bw, inputRowsNum * inputColsNum * sizeof(float));
            current_input_bw_rows = inputRowsNum;
            current_input_bw_cols = inputColsNum;
        }
    
        // Check and allocate memory for p_weights
        if (p_weights_bw == nullptr || current_weights_bw_rows != inputColsNum || current_weights_bw_cols != outputColsNum) {
            hipFree(p_weights_bw);
            hipMalloc(&p_weights_bw, inputColsNum * outputColsNum * sizeof(float));
            current_weights_bw_rows = inputColsNum;
            current_weights_bw_cols = outputColsNum;
        }
    
        // Check and allocate memory for p_d_output
        if (p_d_output_bw == nullptr || current_d_output_bw_rows != inputRowsNum || current_d_output_bw_cols != outputColsNum) {
            hipFree(p_d_output_bw);
            hipMalloc(&p_d_output_bw, inputRowsNum * outputColsNum * sizeof(float));
            current_d_output_bw_rows = inputRowsNum;
            current_d_output_bw_cols = outputColsNum;
        }
    
        // Check and allocate memory for p_d_input
        if (p_d_input_bw == nullptr || current_d_input_bw_rows != inputRowsNum || current_d_input_bw_cols != inputColsNum) {
            hipFree(p_d_input_bw);
            hipMalloc(&p_d_input_bw, inputRowsNum * inputColsNum * sizeof(float));
            current_d_input_bw_rows = inputRowsNum;
            current_d_input_bw_cols = inputColsNum;
        }
    
        // Check and allocate memory for p_d_weights
        if (p_d_weights_bw == nullptr || current_d_weights_bw_rows != inputColsNum || current_d_weights_bw_cols != outputColsNum) {
            hipFree(p_d_weights_bw);
            hipMalloc(&p_d_weights_bw, inputColsNum * outputColsNum * sizeof(float));
            current_d_weights_bw_rows = inputColsNum;
            current_d_weights_bw_cols = outputColsNum;
        }
    
        // Check and allocate memory for p_d_bias
        if (d_bias != nullptr) {
            if (p_d_bias_bw == nullptr || current_d_bias_bw_size != outputColsNum) {
                hipFree(p_d_bias_bw);
                hipMalloc(&p_d_bias_bw, outputColsNum * sizeof(float));
                current_d_bias_bw_size = outputColsNum;
            }
        } else {
            if (p_d_bias_bw != nullptr) {
                hipFree(p_d_bias_bw);
                p_d_bias_bw = nullptr;
                current_d_bias_bw_size = 0;
            }
        }
    
        // Copy data to device
        hipMemcpy(p_input_bw, input, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(p_weights_bw, weights, inputColsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(p_d_output_bw, d_output, inputRowsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);
    
        if (cublas_handle == nullptr) {
            hipblasCreate(&cublas_handle);
        }
    
        const float alf = 1.0f;
        const float bet = 0.0f;
    
        // Compute d_input = d_output * W^T
        hipblasSgemm(cublas_handle,
            HIPBLAS_OP_N,     // No transpose (A is inputColsNum x outputColsNum in column-major)
            HIPBLAS_OP_N,     // No transpose (B is outputColsNum x inputRowsNum in column-major)
            inputColsNum,    // m: rows of op(A) and C (inputColsNum)
            inputRowsNum,    // n: columns of op(B) and C (inputRowsNum)
            outputColsNum,   // k: columns of op(A) and rows of op(B) (outputColsNum)
            &alf,
            p_weights_bw,       // A: inputColsNum x outputColsNum (column-major)
            inputColsNum,    // lda: rows of A in column-major (inputColsNum)
            p_d_output_bw,      // B: outputColsNum x inputRowsNum (column-major)
            outputColsNum,   // ldb: rows of B in column-major (outputColsNum)
            &bet,
            p_d_input_bw,       // C: inputColsNum x inputRowsNum (column-major)
            inputColsNum);   // ldc: rows of C in column-major (inputColsNum)


        // Compute d_weights = X^T * d_output (corrected for row-major)
        hipblasSgemm(cublas_handle,
                    HIPBLAS_OP_N,     // No transpose (A is inputColsNum x inputRowsNum in column-major)
                    HIPBLAS_OP_T,     // Transpose B (B is outputColsNum x inputRowsNum → becomes inputRowsNum x outputColsNum)
                    inputColsNum,    // m: rows of op(A) and C
                    outputColsNum,   // n: columns of op(B) and C
                    inputRowsNum,    // k: columns of op(A) and rows of op(B)
                    &alf,
                    p_input_bw,         // A: inputColsNum x inputRowsNum (column-major)
                    inputColsNum,    // lda: rows of A in column-major (inputColsNum)
                    p_d_output_bw,      // B: outputColsNum x inputRowsNum (column-major)
                    outputColsNum,   // ldb: rows of B in column-major (outputColsNum)
                    &bet,
                    p_d_weights_bw,     // C: inputColsNum x outputColsNum (column-major)
                    inputColsNum);   // ldc: rows of C in column-major (inputColsNum)

        // Compute d_bias = sum(d_output, axis=0) if d_bias is not nullptr
        // Use custom kernel to sum along rows
        if (d_bias != nullptr && p_d_bias_bw != nullptr) {
            int threadsPerBlock = 256;
            int blocksPerGrid = (outputColsNum + threadsPerBlock - 1) / threadsPerBlock;
            sumBiasKernel<<<blocksPerGrid, threadsPerBlock>>>(p_d_bias_bw, p_d_output_bw, inputRowsNum, outputColsNum);
            hipDeviceSynchronize();
        }
    
        // Copy results back to host
        hipMemcpy(d_input, p_d_input_bw, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(d_weights, p_d_weights_bw, inputColsNum * outputColsNum * sizeof(float), hipMemcpyDeviceToHost);
        
        if (d_bias != nullptr && p_d_bias_bw != nullptr) {
            hipMemcpy(d_bias, p_d_bias_bw, outputColsNum * sizeof(float), hipMemcpyDeviceToHost);
        }
    }

    DLLEXPORT void cleanupCudaMemory() {
        // Freeing forward caches
        hipFree(p_input_fw); p_input_fw = nullptr;
        hipFree(p_weights_fw); p_weights_fw = nullptr;
        hipFree(p_output_fw); p_output_fw = nullptr;
        hipFree(p_bias_fw); p_bias_fw = nullptr;
        current_input_fw_rows = current_input_fw_cols = 0;
        current_weights_fw_rows = current_weights_fw_cols = 0;
        current_output_fw_rows = current_output_fw_cols = 0;
        current_bias_fw_size = 0;

        // Freeing backward caches
        hipFree(p_input_bw); p_input_bw = nullptr;
        hipFree(p_weights_bw); p_weights_bw = nullptr;
        hipFree(p_d_output_bw); p_d_output_bw = nullptr;
        hipFree(p_d_input_bw); p_d_input_bw = nullptr;
        hipFree(p_d_weights_bw); p_d_weights_bw = nullptr;
        hipFree(p_d_bias_bw); p_d_bias_bw = nullptr;
        current_input_bw_rows = current_input_bw_cols = 0;
        current_weights_bw_rows = current_weights_bw_cols = 0;
        current_d_output_bw_rows = current_d_output_bw_cols = 0;
        current_d_input_bw_rows = current_d_input_bw_cols = 0;
        current_d_weights_bw_rows = current_d_weights_bw_cols = 0;
        current_d_bias_bw_size = 0;

        // Destroying cuBLAS handle
        if (cublas_handle != nullptr) {
            hipblasDestroy(cublas_handle);
            cublas_handle = nullptr;
        }
    }
}
