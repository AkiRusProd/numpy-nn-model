
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <cstdlib>
#include <iostream>

#ifdef _WIN32
#define DLLEXPORT extern "C" __declspec(dllexport)
#else
#define DLLEXPORT extern "C"
#endif

using namespace std;

// CUDA kernel for adding bias to each column in the output matrix C
// __global__ void addBiasKernel(float *C, const float *bias, int rowsNum, int colsNum) {
//     int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
//     if (idx < colsNum) {
//         for (int i = 0; i < rowsNum; i++) {
//             if (bias != nullptr) { // Check if bias is not null
//                 C[row * colsNum + col] += bias[col];
//             }
//         }
//     }
// }

__global__ void addBiasKernel(float *C, const float *bias, int rowsNum, int colsNum) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < rowsNum && col < colsNum) {
         if (bias != nullptr) {
            C[row * colsNum + col] += bias[col];
        }
    }
}

// CUDA kernel for summing gradients for bias
__global__ void sumBiasKernel(float *d_bias, const float *d_output, int rowsNum, int colsNum) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < colsNum) {
        float sum = 0.0f;
        for (int i = 0; i < rowsNum; i++) {
            sum += d_output[i * colsNum + idx];
        }
        d_bias[idx] = sum;
    }
}

// C(rowsNum, colsNum) = A(rowsNum, width) * B(width, colsNum) + bias
void blasMatMulWithBias(const float *A, const float *B, const float *bias, float *C, const int rowsNum, const int width, const int colsNum) {
    const float alf = 1.0f;
    const float bet = 0.0f;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Matrix multiplication: C = A * B
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, colsNum, rowsNum, width, &alf, B, colsNum, A, width, &bet, C, colsNum);

    // Add bias to each column of C
    if (bias != nullptr) {
        // C(i, j) += bias(j)
        // int threadsPerBlock = 256;  // You can experiment with this value
        // int blocksPerGrid = (rowsNum + threadsPerBlock - 1) / threadsPerBlock;

        dim3 threadsPerBlock(16, 16);
        dim3 blocksPerGrid((rowsNum + 15) / 16, (colsNum + 15) / 16);

        // Add bias to each row for each column (without race conditions)
        addBiasKernel<<<blocksPerGrid, threadsPerBlock>>>(C, bias, rowsNum, colsNum);
        hipDeviceSynchronize();  // Make sure the kernel execution finishes
    }

    hipblasDestroy(handle);
}


extern "C" {
    DLLEXPORT void cudaLinearModuleForward(float *input, float *weights, float *bias, float *output, int inputRowsNum, int inputColsNum, int outputColsNum) {
        float *p_input, *p_weights, *p_bias, *p_output;

        // Device memory allocation
        hipMalloc((void**)&p_input, inputRowsNum * inputColsNum * sizeof(float));
        hipMalloc((void**)&p_weights, inputColsNum * outputColsNum * sizeof(float));
        hipMalloc((void**)&p_output, inputRowsNum * outputColsNum * sizeof(float));

        // Handle bias
        if (bias != nullptr) {
            hipMalloc((void**)&p_bias, outputColsNum * sizeof(float));
            hipMemcpy(p_bias, bias, outputColsNum * sizeof(float), hipMemcpyHostToDevice);
        } else {
            p_bias = nullptr; // Explicitly set to nullptr
        }

        // Copy data to device
        hipMemcpy(p_input, input, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(p_weights, weights, inputColsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);

        // Perform linear layer operation: output = input * weights + bias
        blasMatMulWithBias(p_input, p_weights, p_bias, p_output, inputRowsNum, inputColsNum, outputColsNum);

        // Copy result back to host
        hipMemcpy(output, p_output, inputRowsNum * outputColsNum * sizeof(float), hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(p_input);
        hipFree(p_weights);
        hipFree(p_output);

        if (p_bias != nullptr) {
            hipFree(p_bias);
        }
    }



    DLLEXPORT void cudaLinearModuleBackward(
        float *input, float *weights, float *d_output, 
        float *d_input, float *d_weights, float *d_bias, 
        int inputRowsNum, int inputColsNum, int outputColsNum) {

        float *p_input, *p_weights, *p_d_output, *p_d_input, *p_d_weights, *p_d_bias = nullptr;

        // Device memory allocation
        hipMalloc((void**)&p_input, inputRowsNum * inputColsNum * sizeof(float));
        hipMalloc((void**)&p_weights, inputColsNum * outputColsNum * sizeof(float));
        hipMalloc((void**)&p_d_output, inputRowsNum * outputColsNum * sizeof(float));
        hipMalloc((void**)&p_d_input, inputRowsNum * inputColsNum * sizeof(float));
        hipMalloc((void**)&p_d_weights, inputColsNum * outputColsNum * sizeof(float));

        if (d_bias != nullptr) {
            hipMalloc((void**)&p_d_bias, outputColsNum * sizeof(float));
        }

        // Copy data to device
        hipMemcpy(p_input, input, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(p_weights, weights, inputColsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(p_d_output, d_output, inputRowsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);

        hipblasHandle_t handle;
        hipblasCreate(&handle);

        const float alf = 1.0f;
        const float bet = 0.0f;

        // Compute d_input = d_output * W^T
        hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, inputColsNum, inputRowsNum, outputColsNum,
                    &alf, p_weights, outputColsNum, p_d_output, outputColsNum, &bet, p_d_input, inputColsNum);

        // Compute d_weights = A^T * d_output
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, outputColsNum, inputColsNum, inputRowsNum,
                    &alf, p_d_output, outputColsNum, p_input, inputColsNum, &bet, p_d_weights, outputColsNum);

        // Compute d_bias = sum(d_output, axis=0) if d_bias is not nullptr
        // Use custom kernel to sum along rows
        if (d_bias != nullptr) {
            int threadsPerBlock = 256;
            int blocksPerGrid = (outputColsNum + threadsPerBlock - 1) / threadsPerBlock;
            sumBiasKernel<<<blocksPerGrid, threadsPerBlock>>>(p_d_bias, p_d_output, inputRowsNum, outputColsNum);
            hipDeviceSynchronize();
        }

        // Copy results back to host
        hipMemcpy(d_input, p_d_input, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(d_weights, p_d_weights, inputColsNum * outputColsNum * sizeof(float), hipMemcpyDeviceToHost);

        if (d_bias != nullptr) {
            hipMemcpy(d_bias, p_d_bias, outputColsNum * sizeof(float), hipMemcpyDeviceToHost);
        }

        // Free device memory
        hipFree(p_input);
        hipFree(p_weights);
        hipFree(p_d_output);
        hipFree(p_d_input);
        hipFree(p_d_weights);

        if (p_d_bias != nullptr) {
            hipFree(p_d_bias);
        }

        hipblasDestroy(handle);
    }
}
