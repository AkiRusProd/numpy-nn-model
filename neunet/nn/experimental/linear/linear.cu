
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <cstdlib>
#include <iostream>

#ifdef _WIN32
#define DLLEXPORT extern "C" __declspec(dllexport)
#else
#define DLLEXPORT extern "C"
#endif

using namespace std;

// CUDA kernel for adding bias to each column in the output matrix C
// __global__ void addBiasKernel(float *C, const float *bias, int rowsNum, int colsNum) {
//     int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
//     if (idx < colsNum) {
//         for (int i = 0; i < rowsNum; i++) {
//             if (bias != nullptr) { // Check if bias is not null
//                 C[row * colsNum + col] += bias[col];
//             }
//         }
//     }
// }

__global__ void addBiasKernel(float *C, const float *bias, int rowsNum, int colsNum) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < rowsNum && col < colsNum) {
         if (bias != nullptr) {
            C[row * colsNum + col] += bias[col];
        }
    }
}

// CUDA kernel for summing gradients for bias
__global__ void sumBiasKernel(float *d_bias, const float *d_output, int rowsNum, int colsNum) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < colsNum) {
        float sum = 0.0f;
        for (int i = 0; i < rowsNum; i++) {
            sum += d_output[i * colsNum + idx];
        }
        d_bias[idx] = sum;
    }
}

extern "C" {
    DLLEXPORT void cudaLinearModuleForward(float *input, float *weights, float *bias, float *output, int inputRowsNum, int inputColsNum, int outputColsNum) {
        float *p_input, *p_weights, *p_bias, *p_output;

        // Device memory allocation
        hipMalloc((void**)&p_input, inputRowsNum * inputColsNum * sizeof(float));
        hipMalloc((void**)&p_weights, inputColsNum * outputColsNum * sizeof(float));
        hipMalloc((void**)&p_output, inputRowsNum * outputColsNum * sizeof(float));

        // Handle bias
        if (bias != nullptr) {
            hipMalloc((void**)&p_bias, outputColsNum * sizeof(float));
            hipMemcpy(p_bias, bias, outputColsNum * sizeof(float), hipMemcpyHostToDevice);
        } else {
            p_bias = nullptr; // Explicitly set to nullptr
        }

        // Copy data to device
        hipMemcpy(p_input, input, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(p_weights, weights, inputColsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);

        // Perform linear layer operation: output = input * weights + bias
        // C(rowsNum, colsNum) = A(rowsNum, width) * B(width, colsNum) + bias
        const float alf = 1.0f;
        const float bet = 0.0f;

        hipblasHandle_t handle;
        hipblasCreate(&handle);

        // matrix multiplication: C = A * B^T
        hipblasSgemm(handle,
                    HIPBLAS_OP_T,   // Transpose A (A is originally inputRowsNum x inputColsNum in row-major, becomes inputColsNum x inputRowsNum)
                    HIPBLAS_OP_N,   // No transpose B (B is outputColsNum x inputColsNum in row-major)
                    outputColsNum, // m: rows of op(A) and C (outputColsNum)
                    inputRowsNum,  // n: columns of op(B) and C (inputRowsNum)
                    inputColsNum,  // k: columns of op(A) and rows of op(B) (inputColsNum)
                    &alf,
                    p_weights,     // A: originally B (outputColsNum x inputColsNum in row-major)
                    inputColsNum,  // lda: leading dimension of A (inputColsNum, since row-major)
                    p_input,       // B: originally A (inputRowsNum x inputColsNum in row-major)
                    inputColsNum,  // ldb: leading dimension of B (inputColsNum, since row-major)
                    &bet,
                    p_output,      // C: result matrix (inputRowsNum x outputColsNum in row-major)
                    outputColsNum);// ldc: leading dimension of C (outputColsNum, since row-major)

        // Add bias to each column of C
        if (p_bias != nullptr) {
            // C(i, j) += bias(j)
            dim3 threadsPerBlock(16, 16);
            dim3 blocksPerGrid((inputRowsNum + 15) / 16, (outputColsNum + 15) / 16);

            // Add bias to each row for each column (without race conditions)
            addBiasKernel<<<blocksPerGrid, threadsPerBlock>>>(p_output, p_bias, inputRowsNum, outputColsNum);
            hipDeviceSynchronize();  // Make sure the kernel execution finishes
        }

        hipblasDestroy(handle);

        // Copy result back to host
        hipMemcpy(output, p_output, inputRowsNum * outputColsNum * sizeof(float), hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(p_input);
        hipFree(p_weights);
        hipFree(p_output);

        if (p_bias != nullptr) {
            hipFree(p_bias);
        }
    }



    DLLEXPORT void cudaLinearModuleBackward(
        float *input, float *weights, float *d_output, 
        float *d_input, float *d_weights, float *d_bias, 
        int inputRowsNum, int inputColsNum, int outputColsNum) {

        float *p_input, *p_weights, *p_d_output, *p_d_input, *p_d_weights, *p_d_bias = nullptr;

        // Device memory allocation
        hipMalloc((void**)&p_input, inputRowsNum * inputColsNum * sizeof(float));
        hipMalloc((void**)&p_weights, inputColsNum * outputColsNum * sizeof(float));
        hipMalloc((void**)&p_d_output, inputRowsNum * outputColsNum * sizeof(float));
        hipMalloc((void**)&p_d_input, inputRowsNum * inputColsNum * sizeof(float));
        hipMalloc((void**)&p_d_weights, inputColsNum * outputColsNum * sizeof(float));

        if (d_bias != nullptr) {
            hipMalloc((void**)&p_d_bias, outputColsNum * sizeof(float));
        }

        // Copy data to device
        hipMemcpy(p_input, input, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(p_weights, weights, inputColsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(p_d_output, d_output, inputRowsNum * outputColsNum * sizeof(float), hipMemcpyHostToDevice);

        hipblasHandle_t handle;
        hipblasCreate(&handle);

        const float alf = 1.0f;
        const float bet = 0.0f;

        // Compute d_input = d_output * W^T
        hipblasSgemm(handle,
                    HIPBLAS_OP_N,     // No transpose (A is inputColsNum x outputColsNum in column-major)
                    HIPBLAS_OP_N,     // No transpose (B is outputColsNum x inputRowsNum in column-major)
                    inputColsNum,    // m: rows of op(A) and C (inputColsNum)
                    inputRowsNum,    // n: columns of op(B) and C (inputRowsNum)
                    outputColsNum,   // k: columns of op(A) and rows of op(B) (outputColsNum)
                    &alf,
                    p_weights,       // A: inputColsNum x outputColsNum (column-major)
                    inputColsNum,    // lda: rows of A in column-major (inputColsNum)
                    p_d_output,      // B: outputColsNum x inputRowsNum (column-major)
                    outputColsNum,   // ldb: rows of B in column-major (outputColsNum)
                    &bet,
                    p_d_input,       // C: inputColsNum x inputRowsNum (column-major)
                    inputColsNum);   // ldc: rows of C in column-major (inputColsNum)


        // Compute d_weights = X^T * d_output (corrected for row-major)
        hipblasSgemm(handle,
                    HIPBLAS_OP_N,     // No transpose (A is inputColsNum x inputRowsNum in column-major)
                    HIPBLAS_OP_T,     // Transpose B (B is outputColsNum x inputRowsNum → becomes inputRowsNum x outputColsNum)
                    inputColsNum,    // m: rows of op(A) and C
                    outputColsNum,   // n: columns of op(B) and C
                    inputRowsNum,    // k: columns of op(A) and rows of op(B)
                    &alf,
                    p_input,         // A: inputColsNum x inputRowsNum (column-major)
                    inputColsNum,    // lda: rows of A in column-major (inputColsNum)
                    p_d_output,      // B: outputColsNum x inputRowsNum (column-major)
                    outputColsNum,   // ldb: rows of B in column-major (outputColsNum)
                    &bet,
                    p_d_weights,     // C: inputColsNum x outputColsNum (column-major)
                    inputColsNum);   // ldc: rows of C in column-major (inputColsNum)

        // Compute d_bias = sum(d_output, axis=0) if d_bias is not nullptr
        // Use custom kernel to sum along rows
        if (d_bias != nullptr) {
            int threadsPerBlock = 256;
            int blocksPerGrid = (outputColsNum + threadsPerBlock - 1) / threadsPerBlock;
            sumBiasKernel<<<blocksPerGrid, threadsPerBlock>>>(p_d_bias, p_d_output, inputRowsNum, outputColsNum);
            hipDeviceSynchronize();
        }

        // Copy results back to host
        hipMemcpy(d_input, p_d_input, inputRowsNum * inputColsNum * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(d_weights, p_d_weights, inputColsNum * outputColsNum * sizeof(float), hipMemcpyDeviceToHost);

        if (d_bias != nullptr) {
            hipMemcpy(d_bias, p_d_bias, outputColsNum * sizeof(float), hipMemcpyDeviceToHost);
        }

        // Free device memory
        hipFree(p_input);
        hipFree(p_weights);
        hipFree(p_d_output);
        hipFree(p_d_input);
        hipFree(p_d_weights);

        if (p_d_bias != nullptr) {
            hipFree(p_d_bias);
        }

        hipblasDestroy(handle);
    }
}
